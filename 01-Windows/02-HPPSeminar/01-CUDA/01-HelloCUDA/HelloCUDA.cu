// Header Files
// Standard headers
#include<stdio.h>

// Cuda Headers
#include<hip/hip_runtime.h>

// Global Variables
const int iNumberOfArrayElements = 5;

float* hostInput1 = NULL;
float* hostInput2 = NULL;
float* hostOutput = NULL;

float* deviceInput1 = NULL;
float* deviceInput2 = NULL;
float* deviceOutput = NULL;

// CUDA kernel
__global__ void vecAddGPU(float* in1, float* in2, float* out, int len)
{
    // Code
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < len)
    {
        out[i] = in1[i] + in2[i];
    }
}

// Entry-point Function
int main(void)
{
    // Function Declarations
    void cleanup(void);

    // Variable Declarations
    int size = iNumberOfArrayElements * sizeof(float);
    hipError_t result = hipSuccess;

    // Code
    // Host Memory Allocation
    hostInput1 = (float*)malloc(size);
    if(hostInput1 == NULL)
    {
        printf("Host Memory Allocation is failed for hostinput1 array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }
    hostInput2 = (float*)malloc(size);
    if(hostInput2 == NULL)
    {
        printf("Host Memory Allocation is failed for hostinput2 array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }
    hostOutput = (float*)malloc(size);
    if(hostOutput == NULL)
    {
        printf("Host Memory Allocation is failed for hostOutput array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // Filling values into host arrays
    hostInput1[0] = 101.0;
    hostInput1[1] = 102.0;
    hostInput1[2] = 103.0;
    hostInput1[3] = 104.0;
    hostInput1[4] = 105.0;

    hostInput2[0] = 201.0;
    hostInput2[1] = 202.0;
    hostInput2[2] = 203.0;
    hostInput2[3] = 204.0;
    hostInput2[4] = 205.0;

    // Device memory allocation
    result = hipMalloc((void**)&deviceInput1, size);
    if(result != hipSuccess)
    {
        printf("Device memory allocation is failed for deviceInput1 array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }
    result = hipMalloc((void**)&deviceInput2, size);
    if(result != hipSuccess)
    {
        printf("Device memory allocation is failed for deviceInput2 array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }
    result = hipMalloc((void**)&deviceOutput, size);
    if(result != hipSuccess)
    {
        printf("Device memory allocation is failed for deviceOutput array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // Copy data from host arrays into device arrays
    result = hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
    if(result != hipSuccess)
    {
        printf("Host to Device memory is failed for deviceInput1 array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
    if(result != hipSuccess)
    {
        printf("Host to Device memory is failed for deviceInput2 array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    dim3 dimGrid = dim3(iNumberOfArrayElements, 1, 1);
    dim3 dimBlock = dim3(1, 1, 1);

    // CUDA kernel for vector addition
    vecAddGPU <<<dimGrid, dimBlock >>> (deviceInput1, deviceInput2, deviceOutput, iNumberOfArrayElements);

    // Copy data from device array into host array
    result = hipMemcpy(deviceOutput, hostOutput, size, hipMemcpyHostToDevice);
    if(result != hipSuccess)
    {
        printf("Device to host data copy is failed for hostOutput array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // Vector addition on host
    for (int i=0; i < iNumberOfArrayElements; i++)
    {
        printf("%f + %f\n",hostInput1[i], hostInput2[i], hostOutput[i]);
    }

    // Clean-up
    cleanup();

    return 0;
}

void cleanup(void)
{
    // Code
    if(deviceOutput)
    {
        hipFree(deviceOutput);
        deviceOutput = NULL;
    }
    if(deviceInput2)
    {
        hipFree(deviceInput2);
        deviceInput2 = NULL;
    }
    if(deviceInput1)
    {
        hipFree(deviceInput1);
        deviceInput1 = NULL;
    }
    if(hostOutput)
    {
        free(hostOutput);
        hostOutput = NULL;
    }
    if(hostInput2)
    {
        free(hostInput2);
        hostInput2 = NULL;
    }
    if(hostInput1)
    {
        free(hostInput1);
        hostInput1 = NULL;
    }
    
}
