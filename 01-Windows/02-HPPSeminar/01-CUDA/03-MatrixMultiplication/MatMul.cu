#include "hip/hip_runtime.h"
#include<stdio.h>

// Cuda headers
#include<hip/hip_runtime.h>
#include"helper_timer.h"

// Macros
#define BLOCK_WIDTH 32

// Global Variables
int *hostA = NULL;
int *hostB = NULL;
int *hostC = NULL;
int *gold = NULL;

int *deviceA = NULL;
int *deviceB = NULL;
int *deviceC = NULL;

float timeOnCPU = 0.0f;
float timeOnGPU = 0.0f;

// Cuda Kernel Function
__global__ void matMulGPU(int *A, int *B,int *C, int numARows, int numAColumns, int numBColumns, int numCColumns)
{
    // Variable Declarations
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int column = blockIdx.x * blockDim.x + threadIdx.x;
    // Code
    if((row < numARows) && (column < numBColumns))
    {
        int value = 0.0f;
        for(int k=0; k < numAColumns; k++)
        {
            int a = A[row * numAColumns + k];
            int b = B[k * numBColumns + column];
            value += a*b;
        }
        C[row * numCColumns + column] = value;
    }
}

int main(int argc, char *argv[])
{
    // Function Declarations
    void InitA(int *data, int, int);
    void InitB(int *data, int, int);
    void matMulCPU(int*, int*, int*, int, int, int, int);
    void cleanup(void);

    // Variable Declarations
    int numARows = BLOCK_WIDTH;
    int numAColumns = BLOCK_WIDTH;
    int numBRows = BLOCK_WIDTH;
    int numBColumns = BLOCK_WIDTH;
    int numCRows = numARows;
    int numCColumns = numBColumns;

    int numGoldRows = numARows;
    int numGoldColumns = numBColumns;

    int sizeA = numARows * numAColumns * sizeof(int);
    int sizeB = numBRows * numBColumns * sizeof(int);
    int sizeC = numCRows * numCColumns * sizeof(int);
    int sizeGold = numGoldRows * numGoldColumns * sizeof(int);

    hipError_t result = hipSuccess;

    // Code
    // Host Memory Allocation
    hostA = (int*)malloc(sizeA);
    if(hostA == NULL)
    {
        printf("Host Memory allocation is failed for hostA matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    hostB = (int*)malloc(sizeB);
    if(hostB == NULL)
    {
        printf("Host Memory allocation is failed for hostB matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    hostC = (int*)malloc(sizeC);
    if(hostC == NULL)
    {
        printf("Host Memory allocation is failed for hostC matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    gold = (int*)malloc(sizeGold);
    if(gold == NULL)
    {
        printf("Host Memory allocation is failed for gold matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // Printing matrix dimensions and sizes
    printf("The dimensions of matrix hostA are : %d x %d \n",numARows, numAColumns);
    printf("The dimensions of matrix hostB are : %d x %d \n",numBRows, numBColumns);
    printf("The dimensions of matrix hostC are : %d x %d \n",numCRows, numCColumns);
    printf("The dimensions of matrix gold are : %d x %d \n",numGoldRows, numGoldColumns);

    printf("Size of Matrix hostA = %d\n", sizeA);
    printf("Size of Matrix hostB = %d\n", sizeB);
    printf("Size of Matrix hostC = %d\n", sizeC);

    printf("Size of Matrix gold = %d\n", sizeGold);

    // Fill Source Matrices
    InitA(hostA, numARows, numAColumns);
    InitB(hostB, numBRows, numBColumns);

    // Device memory allocation
    result = hipMalloc((void**)&deviceA, sizeA);
    if(result != hipSuccess)
    {
        printf("Device Memory allocation failed for deviceA matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void**)&deviceB, sizeB);
    if(result != hipSuccess)
    {
        printf("Device Memory allocation failed for deviceB matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void**)&deviceC, sizeC);
    if(result != hipSuccess)
    {
        printf("Device Memory allocation failed for deviceC matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // Copy data from host matrices into device matrices 
    result = hipMemcpy(deviceA , hostA, sizeA, hipMemcpyHostToDevice);
    if(result != hipSuccess)
    {
        printf("Host to device data copy is failed for deiceA matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }
    result = hipMemcpy(deviceB , hostB, sizeB, hipMemcpyHostToDevice);
    if(result != hipSuccess)
    {
        printf("Host to device data copy is failed for deiceA matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }
    
    // CUDA kernel Configuration
    dim3 dimGrid = dim3(ceil((int)numBColumns/(int)BLOCK_WIDTH),ceil((int)numARows/(int)BLOCK_WIDTH), 1);
    dim3 dimBlock = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // CUDA kernel for matrix multiplication
    StopWatchInterface* timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    matMulGPU <<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBColumns, numCColumns);

    sdkStopTimer(&timer);
    timeOnGPU = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
    timer = NULL;

    // Copy data from device matrix into host matrix
    reuslt = hipMemcpy(hostC, deviceC, sizeC, hipMemcpyDeviceToHost);
    if(result != hipSuccess)
    {
        printf("Device to Host data copy is failed for hostC matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // Matrix multiplication on host
    matMulCPU(hostA, hostB, gold, numARows, numAColumns, numBColumns, numCColumns);

    // Comparison
    int breakValue = -1;
    bool bAccuracy = true;
    for (int i = 0; i < numCRows * numCColumns; i++)
    {
        float val1 = gold[i];
        float val2 = hostC[i];
        if(val1 != val2)
        {
            bAccuracy = false;
            breakValue = i;
            break;
        }
    }

    char str[128];
    if(bAccuracy == false)
        sprintf(str, "Comparison of CPU and GPU matrix mult is not within accuracy of 0.000001 at array index %d", breakValue);
    else
        sprintf(str, "Comparison of CPU and GPU matrix mult is within accuracy of 0.000001");
    // Output
    printf("Time taken for Matrix Multiplication on CPU = %.6f\n", timeOnCPU);
    printf("Time taken for Matrix Multiplication on GPU = %.6f\n", timeOnGPU);
    printf("%s",str);

    // Cleanuop
    cleanup();

    return 0;
}

void InitA(int *data, int row, int col)
{
    int num = 1;
    // Code
    for(int i = 0; i < row; i++)
    {
        for(int j=0; j < col; j++)
        {
            *(data + i * col + j) = num;
            num++;
        }
    }
}

void InitB(int *data, int row, int col)
{
    int num = BLOCK_WIDTH;
    // Code
    for(int i = 0; i < row; i++)
    {
        for(int j=0; j < col; j++)
        {
            *(data + i * col + j) = num;
            num--;
        }
    }
}

void matMulCPU(int* A, int* B, int* C, int numARows, int numAColumns, int numBColumns, int numCColumns)
{
    // Code
    StopWatchInterface* timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    for (int i = 0; i < numARows; ++i)
    {
        for(int j = 0; j < numBColumns; ++j)
        {
            int value = 0.0f;
            for (int k = 0; k < numAColumns; ++k)
            {
                int a = A[i * numAColumns + k];
                int b = B[k * numBColumns + j];
                value += a*b;
            }
            C[i * numCColumns + j] = value;
        }
    }
    sdkStopTimer(&timer);
    timeOnCPU = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
    timer = NULL;
}

void cleanup(void)
{
    // Code
    if(deviceC)
    {
        hipFree(deviceC);
        deviceC = NULL;
    }
    if (deviceB)
    {
        hipFree(deviceB);
        deviceB = NULL;
    }
    if (deviceA)
    {
        hipFree(deviceA);
        deviceA = NULL;
    }
    if(gold)
    {
        free(gold);
        gold = NULL;
    }
    if(hostC)
    {
        free(hostC);
        hostC = NULL;
    }
    if(hostB)
    {
        free(hostB);
        hostB = NULL;
    }
    if(hostA)
    {
        free(hostA);
        hostA = NULL;
    }
}
