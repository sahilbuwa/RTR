#include<stdio.h>

int main(void)
{
    // Function Declarations
    void PrintCUDADeviceProperties(void);

    // Code
    PrintCUDADeviceProperties();

}

void PrintCUDADeviceProperties(void)
{
    // Code
    printf("CUDA INFORMATION :\n");
    printf("==============================================================\n");
    hipError_t ret_cuda_rt;
    int dev_count;
    ret_cuda_rt = hipGetDeviceCount(&dev_count);
    if(ret_cuda_rt != hipSuccess)
    {
        printf("CUDA Runtime API Error - hipGetDeviceCount() failed due to %s\n",hipGetErrorString(ret_cuda_rt));
    }
    else if (dev_count == 0)
    {
        printf("There is no cuda supported device on this system.\n");
        return;
    }
    else
    {
        printf("Total Number of cuda supporting GPU device/devices on this system : %d\n", dev_count);
        for(int i=0; i < dev_count; i++)
        {
            hipDeviceProp_t dev_prop;
            int driverVersion = 0, runtimeVersion = 0;

            ret_cuda_rt = hipGetDeviceProperties(&dev_prop, i);
            if(ret_cuda_rt != hipSuccess)
            {
                printf("%s in %s at line %d\n", hipGetErrorString(ret_cuda_rt), __FILE__, __LINE__);
                return;
            }
            printf("\n");
            hipDriverGetVersion(&driverVersion);
            hipRuntimeGetVersion(&runtimeVersion);
            printf("***************CUDA DRIVER AND RUNTIME INFORMATION***********\n");
            printf("=============================================================\n");
            printf("CUDA Driver Version             :%d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10);
            printf("CUDA Runtime Version            :%d.%d\n", runtimeVersion / 1000, (runtimeVersion % 100) / 10);
            printf("\n");
            printf("=============================================================\n");
            printf("*************** GPU Device General Information **************\n");
            printf("=============================================================\n");
            printf("GPU Device Number               :%d\n",i);
            printf("GPU Device Name                 :%s\n", dev_prop.name);
            printf("GPU Device Compute Capability   :%d.%d\n", dev_prop.major, dev_prop.minor);
            printf("GPU Device Clock Rate           :%d\n", dev_prop.clockRate);
            printf("GPU Device Type                 : ");
            if(dev_prop.integrated)
                printf("Integrated (On-Board)\n");
            else
                printf("Discrete ( Card )\n");
            printf("\n");
            printf("********** GPU Device Memory Information***************\n");
            printf("=============================================================\n");
            printf("GPU Device Total Memory         : %.0f GB = %.0f MB = %llu Bytes\n",((float)dev_prop.totalGlobalMem / 1046576.0f) / 1024.0f, (float)dev_prop.totalGlobalMem / 1048576.0f, (unsigned long long) dev_prop.totalGlobalMem);
            printf("GPU Device Constant Memory      :%llu Bytes\n", (unsigned long long)dev_prop.totalConstMem);
            printf("GPU Device Shared Memory Per SMProcessor : %lu\n", (unsigned long)dev_prop.sharedMemPerBlock);
            printf("\n");
            printf("******* GPU DEVICE MULTIPROCESSOR INFORMATION *********\n");
            printf("=============================================================\n");
            printf("GPU Device Number Of SMProcessors   :%d\n", dev_prop.multiProcessorCount);
            printf("GPU Device Number Of Registers Per SMProcessor  :%d\n", dev_prop.regsPerBlock);
            printf("\n");
            printf("***************** GPU DEVICE THREAD INFORMATION*********\n");
            printf("=============================================================\n");
            printf("GPU Device Maximum Number Of Threads Per SMProcessor    : %d\n", dev_prop.maxThreadsPerMultiProcessor);
            printf("GPU Device Maximum Number Of Threads Per Block          :%d\n", dev_prop.maxThreadsPerBlock);
            printf("GPU Device Threads In Warp                              :%d\n", dev_prop.warpSize);
            printf("GPU Device Maximum Thread Dimensions                    :( %d, %d, %d)\n", dev_prop.maxThreadsDim[0], dev_prop.maxThreadsDim[1], dev_prop.maxThreadsDim[2]);
            printf("GPU Device Maximum Grid Dimension                       :( %d, %d, %d)\n", dev_prop.maxGridSize[0], dev_prop.maxGridSize[1], dev_prop.maxGridSize[2]);
            printf("\n");
            printf("**************** GPU DEVICE DRIVER INFORMATION *****************\n");
            printf("=============================================================\n");
            printf("GPU Device has ECC support                              :%s\n", dev_prop.ECCEnabled ? "Enabled" : "Disabled");
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)

            printf("GPU Device Driver Mode ( TCC or WDDM ) :%s\n", dev_prop.tccDriver ? "TCC (Tesla Compute Cluster Driver)" : "WDDM (Windows Display Driver Model )");
#endif
            printf("*****************************************************************************\n");
        }
    }
    
}
